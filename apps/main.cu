#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include "StringUtils.h"
#include "my_cuda_header.cuh"
#include "cuda_class_example.cuh"

#include <stdio.h>
#include <iostream>
#include <vector>

const int MAX_CHAR_ARRAY_SEQUENCE_LENGTH = 20000;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
void addWithCudaSimple(int* c, const int* a, const int* b, unsigned int size);
void addWithCudaParallel(int* c, const int* a, const int* b, unsigned int size); 
// void cudaCompareSequences(int* matchStrengths, std::vector<std::string>& mRNAs, std::vector<std::string>& miRNAs, unsigned int strArrSize);
char** convertArray(std::vector<std::string>&, const int);
int my_strlen(const char*); 

void convert_vec_to_cstrings(char**, std::vector<std::string>, int, int);
void convert_cstrings_to_strings(std::vector<std::string>, char**, int, int);

void convert_vec_to_cstrings(char(*)[MAX_CHAR_ARRAY_SEQUENCE_LENGTH], std::vector<std::string>&, int, int);
void convert_cstrings_to_strings(std::vector<std::string>&, char(*)[MAX_CHAR_ARRAY_SEQUENCE_LENGTH], int, int);

void convert_vec_to_cstrings_v1(char[][MAX_CHAR_ARRAY_SEQUENCE_LENGTH], std::vector<std::string>&, int, int);
void convert_cstrings_to_strings_v1(std::vector<std::string>&, char[][MAX_CHAR_ARRAY_SEQUENCE_LENGTH], int, int);



__global__ void addKernel(int *c, const int *a, const int *b)
{
    printf("Running thread [%d,%d]\n", threadIdx.x, blockIdx.x);
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

/*
 * Call with addKernel_parallel<<<N,1>>> to execute the addKernel_parallel N times
 * in parallel. By using blockIdx.x to index the array, each block handles a different element of the
 * array. Schematic:
 *   - BLOCK 0: c[0] = a[0] + b[0]
 *   - BLOCK 1: c[1] = a[1] + b[1]
 *   - ...
 * In the above calling example, each block executes a single thread.
 */
__global__ void addKernel_parallel(int* c, const int* a, const int* b) {
    printf("Running parallel thread [%d,%d]\n", threadIdx.x, blockIdx.x);
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

/*
__global__ void compareStrings(const char* str1, const char* str2, int len, float* result) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < len) {
        int match = 0;
        for (int i = 0; i < len; i++) {
            if (str1[tid + i] == str2[i]) { // TODO: why not str2[tid+i] ?
                match++;
            }
        }
        result[tid] = (float)match / (float)len;
    }
}

__global__ void compareStringsv1(const char* str1, const char* str2, int len, float* result) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int matches = 0;

    if (tid < len) {
        if (str1[tid] == str2[tid]) {
            matches++;
        }
    }

    __syncthreads(); // this is not recognized by intellisense, but it's valid in cuda

    if (tid == 0) {
        *result = (float)matches / (float)len;
    }


    IMPLEMENTATION IN MAIN()
    const int len = 6;
    char str1[len] = "hello";
    char str2[len] = "world";
    float result;

    compareStringsv1 <<<1, len >> > (str1, str2, len, &result);
    hipDeviceSynchronize();

    std::cout << "The match percentage is: " << result * 100 << "%" << std::endl;
}
*/

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    CudaExampleClass cudaExampleClass(5);
    cudaExampleClass.test_function();

    
    std::vector<std::string> strings1 = { "Hello_\0", "World\0", "123\0" };

    // CHAR[][] FIXED SIZE IMPLEMENTATION
    int strings2_arr_size = 3;
    char strings2[3][10] = { "Hello1", "World1", "1234" };
    char result_strings2[3][10] = {};

    // CHAR[][] WITH INITIALISATION AND INSTANTIATION SEPARATED
    int strings3_arr_size = 3;
    const int char_array_size = 3;
    const int char_element_size = 10;
    // char* strings3 = new char[char_array_size][char_element_size]; // this is not allowed
    // char strings3[char_array_size][char_element_size]; // this is allowed
    // strings3 = {...} // this is NOT allowed
    char strings3[char_array_size][char_element_size] = { "Hey", "It's", "Me!" }; // this is allowed, since it is initalisation
    
    // FIXED SIZE CHAR[][] CHAR-BY-CHAR INITIALISATION AND INSTANTIATION
    char c_strings4[char_array_size][char_element_size];
    // empty array init example
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 10; j++) {
            c_strings4[i][j] = '\0'; // set each element to null character
        }
    }

    // replace the respective indexes with correct chars, other indexes remain set to \0
    for (int i = 0; i < char_array_size; i++) {
        std::string current_string = strings1[i]; // this copies strings from std::vector<std::string> strings1 into char[][]
        
        // this could also be implemented in the following way:
        // char string_as_char_array[char_element_size + 1]; // +1 to account for '\0'
        // strcpy(string_as_char_array, current_string.c_str()); // populate string_as_char array with chars from current_string
        
        // c_strings4[i] = string_as_char_array; // error - cannot modify array; cstrings4[i] returns the reference to the entire char array (entire row)
        
        // assign the correct elements to chars
        for (int j = 0; j < current_string.length(); j++) {
            char c = current_string.at(j);
            c_strings4[i][j] = c;
        }
    
    }

    // convert a char[][] back into std::strings - row by row
    std::vector<std::string> result_strings4 = {};
    for (int i = 0; i < 3; i++) {
        std::string row_str(c_strings4[i], 10);
        result_strings4.push_back(row_str);
    }

    // display strings
    std::cout << "Displaying strings after char[][] testings:" << std::endl;
    for (int i = 0; i < result_strings4.size(); i++) {
        std::cout << "  - " << result_strings4[i] << std::endl;
    }

    // IMPLEMENTATION OF A FUNCTION TO ACCEPT ANY SIZE OF A CHAR ARRAY (attempt 1)
    // create the result char array
    char c_strings5[char_array_size][char_element_size];
    // empty array init example
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 10; j++) {
            c_strings5[i][j] = 'n'; // set each element to null character
        }
    }
    // call the function
    // char** c_strings5_ptr = (char**) c_strings5; // this doesnt work
    char(*c_strings5_ptr)[char_element_size] = c_strings5;
    std::cout << *(*(c_strings5_ptr+1)+2) << std::endl; // access element at row 1, column 2

    // convert_vec_to_cstrings((char**) c_strings5, strings1, char_array_size, char_element_size); // (char**) c_strings5 converts char[][] c_strings5 to a char** pointer of pointers
    // convert the cstrings back into strings
    // std::vector<std::string> result_strings5;
    // convert_cstrings_to_strings(result_strings5, (char**)c_strings5, char_array_size, char_element_size);

    // IMPLEMENTATION OF A FUNCTION TO ACCEPT ANY SIZE OF A CHAR ARRAY (attempt 2)
    char c_strings6[char_array_size][MAX_CHAR_ARRAY_SEQUENCE_LENGTH];
    StringUtils::init_Cstrings_array(c_strings6, char_array_size);
    convert_vec_to_cstrings(c_strings6, strings1, char_array_size, MAX_CHAR_ARRAY_SEQUENCE_LENGTH);
    // convert cstrings back into strings
    std::vector<std::string> result_strings6;
    convert_cstrings_to_strings(result_strings6, c_strings6, char_array_size, MAX_CHAR_ARRAY_SEQUENCE_LENGTH);

    // display strings
    std::cout << "Displaying strings after char[][] function test for c_strings6:" << std::endl;
    for (int i = 0; i < result_strings6.size(); i++) {
        std::cout << "  - " << result_strings6[i] << std::endl;
    }

    // IMPLEMENTATION OF A FUNCTION TO ACCEPT ANY SIZE OF A CHAR ARRAY V2 (without using pointers)
    char c_strings7[char_array_size][MAX_CHAR_ARRAY_SEQUENCE_LENGTH];
    StringUtils::init_Cstrings_array(c_strings7, char_array_size);
    // empty array init example
    // for (int i = 0; i < 3; i++) {
    //    for (int j = 0; j < MAX_CHAR_ARRAY_SEQUENCE_LENGTH; j++) {
    //        c_strings7[i][j] = '\0'; // set each element to null character
    //    }
    // }
    
    // convert to cstrings
    // convert_vec_to_cstrings_v1(c_strings7, strings1, char_array_size, MAX_CHAR_ARRAY_SEQUENCE_LENGTH);
    StringUtils::convert_strings_to_Cstrings_ptr(c_strings7, strings1, char_array_size, MAX_CHAR_ARRAY_SEQUENCE_LENGTH);
    // convert cstrings back into strings
    std::vector<std::string> result_strings7;
    // convert_cstrings_to_strings_v1(result_strings7, c_strings7, char_array_size, MAX_CHAR_ARRAY_SEQUENCE_LENGTH);
    StringUtils::convert_Cstrings_to_strings_ptr(result_strings7, c_strings7, char_array_size, MAX_CHAR_ARRAY_SEQUENCE_LENGTH);
    // display strings
    std::cout << "Displaying strings after char[][] POINTERLESS function test:" << std::endl;
    for (int i = 0; i < result_strings7.size(); i++) {
        std::cout << "  - " << result_strings7[i] << std::endl;
    }

    std::vector<char(*)[10]> vec; // char(*)[10] is a pointer to a character array !!!
    for (int i = 0; i < strings2_arr_size; i++) {
        vec.push_back(&strings2[i]);
    }
    
    // populate cstrings with c-style strings from strings1 (vector of std::string)
    const char** cstrings = new const char*[strings1.size()]; // each element is a pointer to a C-style string
    for (int i = 0; i < strings1.size(); i++) {
        cstrings[i] = strings1[i].c_str();
    }
    std::cout << "Displaying cstrings" << std::endl;
    for (int i = 0; i < strings1.size(); i++) {
        std::cout << cstrings[i] << std::endl;
    }

    const char** result_cstrings = new const char*[strings1.size()];
    cudaExampleClass.test_char_copy_function(result_cstrings, cstrings, strings1.size());
    hipDeviceSynchronize();
    std::cout << "Printing c-style strings sent to and copied back from the gpu:" << std::endl;
    for (int i = 0; i < strings1.size(); i++) {
        std::cout << "  - " << result_cstrings[i] << std::endl;
    }

    // TODO: WORK FROM HERE !!!!!
    cudaExampleClass.test_char_copy_function_v3(result_strings2, strings2, strings2_arr_size);
    std::cout << "Test_char_copy_function_v3 results:" << std::endl;
    std::cout << "  - " << result_strings2[0] << std::endl;
    std::cout << "  - " << result_strings2[1] << std::endl;
    std::cout << "  - " << result_strings2[2] << std::endl;

    // test my_strlen implementation
    std::cout << "Strlen implentation my_strlen result for 'Hello' is: " << my_strlen("Hello") << std::endl;

    // Add vectors in parallel.
    /* hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    */

    addWithCudaSimple(c, a, b, arraySize);
    // addWithCudaParallel(c, a, b, arraySize);
    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    return 0;
}

/*
 * It's not possible to have an unspecified size for the second dimension of a multidimensional array when passing it as a function
 * parameter. When you declare a multidimensional array with an empty second dimension, such as char input_char_array[][ ], the compiler 
 * does not know how much memory to allocate for each row.
 * 
 * Example of an incorrect function: void convert_vec_to_cstrings(char input_char_array[][]) {...}
 * 
 * Instead, you can use a pointer to a dynamically allocated array and pass it as a function parameter.
 * 
 * TODO: This is not allowed with an error "Expression must a pointer to a complete object type".
 */
void convert_vec_to_cstrings(char(*input_char_array)[], std::vector<std::string> strings) {
    std::cout << "THIS FUNCTION HAS NO FUNCTIONALIY" << std::endl;
    return; // todo: remove this if you resolve the error

    int strings_size = strings.size();
    // todo: check if size of input_char_array (the first dimensiuon) and strings_size match
    for (int i = 0; i < strings_size; i++) {
        std::string current_string = strings[i]; // this copies strings from std::vector<std::string> strings1 into char[][]

        // assign the correct elements to chars
        for (int j = 0; j < current_string.length(); j++) {
            char c = current_string.at(j);
            // input_char_array[i][j] = c; // error here
        }

    }
}

// THIS DOESNT WORK, memory access violation. You need to pass char(*input_char_array_ptr)[] and then access values by *(*input_char_array_ptr+i)+j);
void convert_vec_to_cstrings(char** input_char_array, std::vector<std::string> strings, int row_count, int col_count) {
    std::cout << "THIS FUNCTION HAS NO FUNCTIONALIY" << std::endl;
    return; // todo: remove this if you resolve the error

    int strings_size = strings.size();
    // todo: check if size of input_char_array (the first dimensiuon) = row_count and strings_size match
    for (int i = 0; i < strings_size; i++) {
        std::string current_string = strings[i];

        int string_length = current_string.length();
        // todo: check if the size of string_length is less than the second dimension of input_char_array (represented by col_count)
        for (int j = 0; j < string_length; j++) {
            char c = current_string.at(j);
            input_char_array[i][j] = c;
        }
    }
}

// THIS DOESNT WORK, memory access violation. You need to pass char(*input_char_array_ptr)[] and then access values by *(*input_char_array_ptr+i)+j);
void convert_cstrings_to_strings(std::vector<std::string> dst_strings, char** input_char_array, int row_count, int col_count) {
    std::cout << "THIS FUNCTION HAS NO FUNCTIONALIY" << std::endl;
    return; // todo: remove this if you resolve the error
    // convert a char[][] back into std::strings - row by row
    for (int i = 0; i < row_count; i++) {
        std::string row_str(input_char_array[i], col_count);
        dst_strings.push_back(row_str);
    }
}

// THIS WORKS
void convert_vec_to_cstrings(char(*input_char_array_ptr)[MAX_CHAR_ARRAY_SEQUENCE_LENGTH], std::vector<std::string> &strings, int row_count, int col_count) {
    int strings_size = strings.size();
    // todo: check if size of input_char_array (the first dimensiuon) = row_count and strings_size match
    for (int i = 0; i < strings_size; i++) {
        std::string current_string = strings[i];
        int string_length = current_string.length();
        // todo: check if the size of string_length is less than the second dimension of input_char_array (represented by col_count)

        for (int j = 0; j < string_length; j++) {
            char c = current_string.at(j);
            *(*(input_char_array_ptr + i) + j) = c;
        }

    }
}

// THIS WORKS
// note: std::vector should be passed by reference (default it is passed by value)! 
void convert_cstrings_to_strings(std::vector<std::string> &dst_strings, char(*input_char_array_ptr)[MAX_CHAR_ARRAY_SEQUENCE_LENGTH], int row_count, int col_count) {
    for (int i = 0; i < row_count; i++) {
        std::string row_str(*(input_char_array_ptr + i), col_count);
        dst_strings.push_back(row_str);
    }
}

// note: std::vector should be passed by reference (default it is passed by value)! 
void convert_vec_to_cstrings_v1(char input_char_array[][MAX_CHAR_ARRAY_SEQUENCE_LENGTH], std::vector<std::string> &strings, int row_count, int col_count) {
    int strings_size = strings.size();
    // todo: check if size of input_char_array (the first dimensiuon) = row_count and strings_size match
    for (int i = 0; i < strings_size; i++) {
        std::string current_string = strings[i];
        int string_length = current_string.length();
        // todo: check if string_length < col_count
        for (int j = 0; j < string_length; j++) {
            char c = current_string.at(j);
            input_char_array[i][j] = c;
        }
    }
}

// note: std::vector should be passed by reference (default it is passed by value)! 
void convert_cstrings_to_strings_v1(std::vector<std::string> &dst_strings, char input_char_array[][MAX_CHAR_ARRAY_SEQUENCE_LENGTH], int row_count, int col_count) {
    for (int i = 0; i < row_count; i++) {
        std::string row_str(input_char_array[i], col_count);
        dst_strings.push_back(row_str);
    }
}

/*
void cudaCompareSequences(int* matchStrengths, std::vector<std::string>& mRNAs, std::vector<std::string>& miRNAs, unsigned int strArrSize) {
    // TODO: pass two different sizes, one for mRNAs and one for miRNAs
    // TODO: CONVERT ALL STD::STRING TO CHAR ARRAYS, SINCE CUDA DOESNT ACCEPT STD::STRING
    
    // convert std::string to const char, since CUDA doesn't accept std::string
    char** mRNAs_charArr = convertArray(mRNAs, mRNAs.size());
    char** miRNAs_charArr = convertArray(miRNAs, miRNAs.size());

    // declare device (GPU) copies
    char** dev_mRNAs;
    char** dev_miRNAs;
    int* dev_matchStrengths = 0;

    // set primaty gpu
    hipSetDevice(0);

    // allocate gpu memory
    hipMalloc((void**)&dev_mRNAs, strArrSize * sizeof(char));
    hipMalloc((void**)&dev_miRNAs, strArrSize * sizeof(char));
    hipMalloc((void**)&dev_matchStrengths, strArrSize * sizeof(int)); // todo: here, calculate size by: mRNAs_size * miRNAs_size * sizeof(int)

    // copy data to device
    hipMemcpy(dev_mRNAs, mRNAs_charArr, strArrSize * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(dev_miRNAs, miRNAs_charArr, strArrSize * sizeof(char), hipMemcpyHostToDevice);

    // run the GPU kernel
    stringCompareKernel_parallel<<<strArrSize, 1>>> (dev_matchStrengths, dev_mRNAs, dev_miRNAs);

    // copy result back to host
    hipMemcpy(matchStrengths, dev_matchStrengths, strArrSize * sizeof(int), hipMemcpyDeviceToHost);

    // free device memory
    hipFree(dev_matchStrengths);
    hipFree(dev_miRNAs);
    hipFree(dev_mRNAs);

    return;
}
*/


// Function to add vectors a and b into c, without the error checks
void addWithCudaSimple(int* c, const int* a, const int* b, unsigned int size) {
    // declare device (GPU) copies
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;

    // set the primary GPU (change this on multi-gpu devices)
    hipSetDevice(0);

    // allocate GPU memory
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));
    hipMalloc((void**)&dev_c, size * sizeof(int));

    // copy data from host to device
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // run the gpu kernel
    addKernel<<<1,size>>>(dev_c, dev_a, dev_b);

    // test the gpu kernel from .cu / .cuh separation
    my_kernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // copy result back to host
    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    // free device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return;
}

void addWithCudaParallel(int* c, const int* a, const int* b, unsigned int size) {
    // declare device (GPU) copies
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;

    // set the primary GPU (change this on multi-gpu devices)
    hipSetDevice(0);

    // allocate GPU memory
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));
    hipMalloc((void**)&dev_c, size * sizeof(int));

    // copy data from host to device
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // run the gpu kernel
    addKernel_parallel<<<size, 1>>>(dev_c, dev_a, dev_b);

    // copy result back to host
    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    // free device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return;
}

// Helper function for using CUDA to add vectors in parallel.
// This function also has error-checking functionality at each step of the way.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    // device (GPU) copies of a, b and c input params
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1,size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

/*
 * Return type is char** (double pointer) as we are returning an array of char* ponters.
 * 
 */
char** convertArray(std::vector<std::string>& string_array, const int arraySize) {
    // create new array to hold the char arrays
    char** result = new char*[arraySize];

    // convert each element
    for (int i = 0; i < arraySize; i++) {
        const char* str = string_array[i].c_str();
        result[i] = new char[strlen(str) + 1];
        strcpy(result[i], str);
    }

    // return the array of char arrays
    return result;
}

/*
 * A test implementation of the std::strlen function to be used in the CUDA kernel.
 */
int my_strlen(const char* cstr) {
    int len = 0;
    while (cstr[len] != '\0') { // !! WARNING !!: using "\0" produces an error as it is regarded as const char*, whereas using '\0' is regarded as char
        len++;
    }
    return len;
}


